/* CUDA 10.2 has a bug that prevents including <cuda/atomic> from two separate
 * object files. As a workaround, we include ex2.cu directly here. */
#include "ex2.cu"

#include <cassert>
#include <iostream>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>

#include <infiniband/verbs.h>

class server_rpc_context : public rdma_server_context {
private:
    std::unique_ptr<gpu_image_processing_context> gpu_context;

public:
    explicit server_rpc_context(uint16_t tcp_port) : rdma_server_context(tcp_port),
        gpu_context(create_queues_server(256))
    {
    }

    virtual void event_loop() override
    {
        /* so the protocol goes like this:
         * 1. we'll wait for a CQE indicating that we got an Send request from the client.
         *    this tells us we have new work to do. The wr_id we used in post_recv tells us
         *    where the request is.
         * 2. now we send an RDMA Read to the client to retrieve the request.
         *    we will get a completion indicating the read has completed.
         * 3. we process the request on the GPU.
         * 4. upon completion, we send an RDMA Write with immediate to the client with
         *    the results.
         */
        rpc_request* req;
        uchar *img_in;
        uchar *img_out;

        bool terminate = false, got_last_cqe = false;

        while (!terminate || !got_last_cqe) {
            // Step 1: Poll for CQE
            struct ibv_wc wc;
            int ncqes = ibv_poll_cq(cq, 1, &wc);
            if (ncqes < 0) {
                perror("ibv_poll_cq() failed");
                exit(1);
            }
            if (ncqes > 0) {
		VERBS_WC_CHECK(wc);

                switch (wc.opcode) {
                case IBV_WC_RECV:
                    /* Received a new request from the client */
                    req = &requests[wc.wr_id];
                    img_in = &images_in[wc.wr_id * IMG_SZ];

                    /* Terminate signal */
                    if (req->request_id == -1) {
                        printf("Terminating...\n");
                        terminate = true;
                        goto send_rdma_write;
                    }

                    /* Step 2: send RDMA Read to client to read the input */
                    post_rdma_read(
                        img_in,             // local_src
                        req->input_length,  // len
                        mr_images_in->lkey, // lkey
                        req->input_addr,    // remote_dst
                        req->input_rkey,    // rkey
                        wc.wr_id);          // wr_id
                    break;

                case IBV_WC_RDMA_READ:
                    /* Completed RDMA read for a request */
                    req = &requests[wc.wr_id];
                    img_in = &images_in[wc.wr_id * IMG_SZ];
                    img_out = &images_out[wc.wr_id * IMG_SZ];

                    // Step 3: Process on GPU
                    gpu_context->enqueue(wc.wr_id, img_in, img_out);
					break;
                    
                case IBV_WC_RDMA_WRITE:
                    /* Completed RDMA Write - reuse buffers for receiving the next requests */
                    post_recv(wc.wr_id % OUTSTANDING_REQUESTS);

		    if (terminate)
			got_last_cqe = true;

                    break;
                default:
                    printf("Unexpected completion\n");
                    assert(false);
                }
            }

            // Dequeue completed GPU tasks
            int dequeued_img_id;
            if (gpu_context->dequeue(&dequeued_img_id)) {
                req = &requests[dequeued_img_id];
                img_out = &images_out[dequeued_img_id * IMG_SZ];

send_rdma_write:
                // Step 4: Send RDMA Write with immediate to client with the response
				post_rdma_write(
                    req->output_addr,                       // remote_dst
                    terminate ? 0 : req->output_length,     // len
                    req->output_rkey,                       // rkey
                    terminate ? 0 : img_out,                // local_src
                    mr_images_out->lkey,                    // lkey
                    dequeued_img_id + OUTSTANDING_REQUESTS, // wr_id
                    (uint32_t*)&req->request_id);           // immediate
            }
        }
    }
};

class client_rpc_context : public rdma_client_context {
private:
    uint32_t requests_sent = 0;
    uint32_t send_cqes_received = 0;

    struct ibv_mr *mr_images_in; /* Memory region for input images */
    struct ibv_mr *mr_images_out; /* Memory region for output images */
public:
    explicit client_rpc_context(uint16_t tcp_port) : rdma_client_context(tcp_port)
    {
    }

    ~client_rpc_context()
    {
        kill();
    }

    virtual void set_input_images(uchar *images_in, size_t bytes) override
    {
        /* register a memory region for the input images. */
        mr_images_in = ibv_reg_mr(pd, images_in, bytes, IBV_ACCESS_REMOTE_READ);
        if (!mr_images_in) {
            perror("ibv_reg_mr() failed for input images");
            exit(1);
        }
    }

    virtual void set_output_images(uchar *images_out, size_t bytes) override
    {
        /* register a memory region for the output images. */
        mr_images_out = ibv_reg_mr(pd, images_out, bytes, IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_WRITE);
        if (!mr_images_out) {
            perror("ibv_reg_mr() failed for output images");
            exit(1);
        }
    }

    virtual bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        if (requests_sent - send_cqes_received == OUTSTANDING_REQUESTS)
            return false;

        struct ibv_sge sg; /* scatter/gather element */
        struct ibv_send_wr wr; /* WQE */
        struct ibv_send_wr *bad_wr; /* ibv_post_send() reports bad WQEs here */

        /* step 1: send request to server using Send operation */
        
        struct rpc_request *req = &requests[requests_sent % OUTSTANDING_REQUESTS];
        req->request_id = img_id;
        req->input_rkey = img_in ? mr_images_in->rkey : 0;
        req->input_addr = (uintptr_t)img_in;
        req->input_length = IMG_SZ;
        req->output_rkey = img_out ? mr_images_out->rkey : 0;
        req->output_addr = (uintptr_t)img_out;
        req->output_length = IMG_SZ;

        /* RDMA send needs a gather element (local buffer)*/
        memset(&sg, 0, sizeof(struct ibv_sge));
        sg.addr = (uintptr_t)req;
        sg.length = sizeof(*req);
        sg.lkey = mr_requests->lkey;

        /* WQE */
        memset(&wr, 0, sizeof(struct ibv_send_wr));
        wr.wr_id = img_id; /* helps identify the WQE */
        wr.sg_list = &sg;
        wr.num_sge = 1;
        wr.opcode = IBV_WR_SEND;
        wr.send_flags = IBV_SEND_SIGNALED; /* always set this in this excersize. generates CQE */

        /* post the WQE to the HCA to execute it */
        if (ibv_post_send(qp, &wr, &bad_wr)) {
            perror("ibv_post_send() failed");
            exit(1);
        }

        ++requests_sent;

        return true;
    }

    virtual bool dequeue(int *img_id) override
    {
        /* When WQE is completed we expect a CQE */
        /* We also expect a completion of the RDMA Write with immediate operation from the server to us */
        /* The order between the two is not guarenteed */

        struct ibv_wc wc; /* CQE */
        int ncqes = ibv_poll_cq(cq, 1, &wc);
        if (ncqes < 0) {
            perror("ibv_poll_cq() failed");
            exit(1);
        }
        if (ncqes == 0)
            return false;

	VERBS_WC_CHECK(wc);

        switch (wc.opcode) {
        case IBV_WC_SEND:
            ++send_cqes_received;
            return false;
        case IBV_WC_RECV_RDMA_WITH_IMM:
            *img_id = wc.imm_data;
            break;
        default:
            printf("Unexpected completion type\n");
            assert(0);
        }

        /* step 2: post receive buffer for the next RPC call (next RDMA write with imm) */
        post_recv();

        return true;
    }

    void kill()
    {
        while (!enqueue(-1, // Indicate termination
                       NULL, NULL)) ;
        int img_id = 0;
        bool dequeued;
        do {
            dequeued = dequeue(&img_id);
        } while (!dequeued || img_id != -1);
    }
};


class server_queues_context : public rdma_server_context {
private:
    queues_gpu_context gpu_context;
    int blocks;
    /* TODO: add memory region(s) for CPU-GPU queues */
    struct ibv_mr *mr_cpu_to_gpu; 
    struct ibv_mr *mr_gpu_to_cpu; 
    queue<cpu_to_gpu_entry> *cpu_to_gpu;
    queue<gpu_to_cpu_entry> *gpu_to_cpu;
    
public:
    explicit server_queues_context(uint16_t tcp_port) : 
        rdma_server_context(tcp_port),
        gpu_context(queues_gpu_context(256)){
        
        /* TODO Initialize additional server MRs as needed. */
        blocks = gpu_context.getBlocks();
        gpu_context.getQueues(&cpu_to_gpu , &gpu_to_cpu); //get pointers to queues
        
        // register the memory regions
        mr_cpu_to_gpu = ibv_reg_mr(pd, cpu_to_gpu, sizeof(queue<cpu_to_gpu_entry>[blocks]) , IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_WRITE | IBV_ACCESS_REMOTE_READ);
		//std::cout << "cpu_to_gpu address (second):" << cpu_to_gpu << std::endl;
        if (!mr_cpu_to_gpu) {
            perror("ibv_reg_mr() failed for mr_cpu_to_gpu");
            exit(1);
        }
        mr_gpu_to_cpu = ibv_reg_mr(pd, gpu_to_cpu, sizeof(queue<gpu_to_cpu_entry>[blocks]) , IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_WRITE | IBV_ACCESS_REMOTE_READ);
        if (!mr_gpu_to_cpu) {
            perror("ibv_reg_mr() failed for mr_gpu_to_cpu");
            exit(1);
        }

        struct rpc_request connectionContext[2];
        
        connectionContext[0].request_id = blocks;
        connectionContext[0].input_rkey = mr_images_in->rkey;
        connectionContext[0].input_length = OUTSTANDING_REQUESTS * IMG_SZ;
        connectionContext[0].input_addr = (uint64_t) images_in;
        connectionContext[0].output_rkey = mr_images_out->rkey;
        connectionContext[0].output_length = OUTSTANDING_REQUESTS * IMG_SZ;
        connectionContext[0].output_addr = (uint64_t) images_out;
        connectionContext[1].request_id = blocks;
        connectionContext[1].input_rkey = mr_cpu_to_gpu->rkey;
        connectionContext[1].input_length = sizeof(queue<cpu_to_gpu_entry>[blocks]);
        connectionContext[1].input_addr = (uint64_t) cpu_to_gpu;
        connectionContext[1].output_rkey = mr_gpu_to_cpu->rkey;
        connectionContext[1].output_length = sizeof(queue<gpu_to_cpu_entry>[blocks]);
        connectionContext[1].output_addr = (uint64_t) gpu_to_cpu;

        /* TODO Exchange rkeys, addresses, and necessary information (e.g.
         * number of queues) with the client */
         send_over_socket(connectionContext, 2 * sizeof(rpc_request));
        
    }

    ~server_queues_context(){
        /* TODO destroy the additional server MRs here */
        ibv_dereg_mr(mr_cpu_to_gpu);
        ibv_dereg_mr(mr_gpu_to_cpu);
    }

    virtual void event_loop() override{
        /* TODO simplified version of server_rpc_context::event_loop. As the
         * client use one sided operations, we only need one kind of message to
         * terminate the server at the end. */
        
		bool terminate = false;
		rpc_request* req;
		
        while (!terminate) {
            // Step 1: Poll for CQE
            struct ibv_wc wc;
            int ncqes = ibv_poll_cq(cq, 1, &wc);
            if (ncqes < 0) {
                perror("ibv_poll_cq() failed");
                exit(1);
            }
            if (ncqes > 0) {
		        VERBS_WC_CHECK(wc);
                switch (wc.opcode) {
                case IBV_WC_RECV:
                    /* Received a new request from the client */
                    req = &requests[0];

                    /* Terminate signal */
                    if (req->request_id == -1) {
                        printf("Terminating...\n");
                        terminate = true;
                    }
                    break;
                default:
                    printf("Unexpected completion\n");
                    assert(false);
                }
            }
		}
    }
};
    
struct queue_context{
    int pi;
    int ci;
    cpu_to_gpu_entry c2g;
    gpu_to_cpu_entry g2c;
};

class client_queues_context : public rdma_client_context {
private:
    /* TODO add necessary context to track the client side of the GPU's
     * producer/consumer queues */
	
	uint32_t requests_enqueued = 0;
    uint32_t requests_dequeued = 0;
	uchar* out_images;
    int blocks;
    int producer_nextBlockIdx = 0, consumer_nextBlockIdx = 0;
    struct queue_context q_context;
    struct rpc_request connectionContext[2];
    struct ibv_mr *mr_images_in; /* Memory region for input images */
    struct ibv_mr *mr_images_out; /* Memory region for output images */
    struct ibv_mr *mr_queue_context;
    /* TODO define other memory regions used by the client here */

public:
    client_queues_context(uint16_t tcp_port) : rdma_client_context(tcp_port)
    {
        /* TODO communicate with server to discover number of queues, necessary
         * rkeys / address, or other additional information needed to operate
         * the GPU queues remotely. */
        queue<cpu_to_gpu_entry> c2gTempQueue;
        queue<gpu_to_cpu_entry> g2cTempQueue;

        recv_over_socket(connectionContext, 2 * sizeof(rpc_request));
        blocks = connectionContext[0].request_id;
        mr_queue_context = ibv_reg_mr(pd, &q_context, sizeof(queue_context), IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_WRITE | IBV_ACCESS_REMOTE_READ);
        if (!mr_queue_context) {
            perror("ibv_reg_mr() failed for input images");
            exit(1);
        }
    }

    ~client_queues_context()
    {
	/* TODO terminate the server and release memory regions and other resources */
        ibv_dereg_mr(mr_queue_context);
    }

    virtual void set_input_images(uchar *images_in, size_t bytes) override
    {
        // TODO register memory
        /* register a memory region for the input images. */
        mr_images_in = ibv_reg_mr(pd, images_in, bytes, IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_WRITE | IBV_ACCESS_REMOTE_READ);
        if (!mr_images_in) {
            perror("ibv_reg_mr() failed for input images");
            exit(1);
        }
    }

    virtual void set_output_images(uchar *images_out, size_t bytes) override
    {
        // TODO register memory
        /* register a memory region for the output images. */
		out_images = images_out;
        mr_images_out = ibv_reg_mr(pd, images_out, bytes, IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_WRITE | IBV_ACCESS_REMOTE_READ);
        if (!mr_images_out) {
            perror("ibv_reg_mr() failed for output images");
            exit(1);
        }
    }

    virtual bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        /* TODO use RDMA Write and RDMA Read operations to enqueue the task on
         * a CPU-GPU producer consumer queue running on the server. */
        
		if (requests_enqueued - requests_dequeued == OUTSTANDING_REQUESTS)
            return false;

		uint32_t rkey = connectionContext[1].input_rkey;
        int num_of_used_slots = 0 , piV = 0 , ciV = 0;
		queue<cpu_to_gpu_entry>* queue_ptr = (queue<cpu_to_gpu_entry>*) (connectionContext[1].input_addr);
        for(int count = 0; count < blocks ; count++ , producer_nextBlockIdx = ((producer_nextBlockIdx + 1) % blocks) ){
            num_of_used_slots = queueStatus((uint64_t)&(queue_ptr[producer_nextBlockIdx].pi), (uint64_t)&(queue_ptr[producer_nextBlockIdx].ci), rkey , &piV , &ciV);
            if(num_of_used_slots < NSLOTS) break;
        }
        if(num_of_used_slots == NSLOTS) return false;
        write_image(&(queue_ptr[producer_nextBlockIdx]), img_id, img_in, img_out, piV);
		producer_nextBlockIdx = ((producer_nextBlockIdx + 1) % blocks);
		
		++requests_enqueued;
		std::cout << "enqueued:" << requests_enqueued << std::endl;
        return true;
    }

    virtual bool dequeue(int *img_id) override
    {
        /* TODO use RDMA Write and RDMA Read operations to detect the completion and dequeue a processed image
         * through a CPU-GPU producer consumer queue running on the server. */
        
        uint32_t rkey = connectionContext[1].output_rkey;
        int num_of_used_slots = 0 , piV = 0 , ciV = 0;
		queue<gpu_to_cpu_entry>* queue_ptr = (queue<gpu_to_cpu_entry>*) (connectionContext[1].output_addr);
        for(int count = 0; count < blocks ; count++ , consumer_nextBlockIdx = ((consumer_nextBlockIdx + 1) % blocks) ){
            num_of_used_slots = queueStatus((uint64_t)&(queue_ptr[consumer_nextBlockIdx].pi), (uint64_t)&(queue_ptr[consumer_nextBlockIdx].ci), rkey , &piV , &ciV);
            if(num_of_used_slots < NSLOTS) break;
        }
        if(num_of_used_slots == 0) return false;
        read_image(&(queue_ptr[consumer_nextBlockIdx]), img_id, ciV);
		consumer_nextBlockIdx = ((consumer_nextBlockIdx + 1) % blocks);
		
		++requests_dequeued;
		std::cout << "dequeued:" << requests_dequeued << std::endl;
        return true;
    }
    

    int queueStatus(uint64_t pi_ptr, uint64_t ci_ptr, uint32_t rkey, int *piVal , int *ciVal){
        int pi = 0 , ci = 0;

        //rdma read pi
        post_rdma_read(
            &q_context.pi,              // local_src
            sizeof(int),                // len
            mr_queue_context->lkey,     // lkey
            pi_ptr,                // remote_dst
            rkey,                       // rkey
            1);                         // wr_id
        
        //rdma read ci
        post_rdma_read(
            &q_context.ci,              // local_src
            sizeof(int),                // len
            mr_queue_context->lkey,     // lkey
            ci_ptr,                // remote_dst
            rkey,                       // rkey
            2);                         // wr_id
        
        bool pi_rcv = false , ci_rcv = false;
        
        while (!pi_rcv || !ci_rcv) {
            struct ibv_wc wc;
            int ncqes = ibv_poll_cq(cq, 1, &wc);
            if (ncqes < 0) {
                perror("ibv_poll_cq() failed");
                exit(1);
            }
            if (ncqes > 0) {
                VERBS_WC_CHECK(wc);
                if (wc.opcode == IBV_WC_RDMA_READ) {
                    if(wc.wr_id == 1) {
                        pi = q_context.pi;
                        pi_rcv = true;
                    }
                    if(wc.wr_id == 2){
                         ci = q_context.ci;
                         ci_rcv = true;
                    } 
                }
                else{
                    printf("Unexpected completion\n");
                    assert(false);
                }
            }
        }
        *piVal = pi;
        *ciVal = ci;
        return pi - ci;
    }

    void write_image(queue<cpu_to_gpu_entry>* queue_ptr, int img_id, uchar *img_in, uchar *img_out , int pi){
        
		uint64_t remote_dst = connectionContext[0].input_addr + IMG_SZ * (img_id % OUTSTANDING_REQUESTS);
        uint32_t rkey = connectionContext[0].input_rkey;
        
		//Write image data
        post_rdma_write(remote_dst, IMG_SZ , rkey, img_in, mr_images_in->lkey, 0, nullptr);
        
		//post image to queue:
        //a. update data entry
        q_context.c2g.img_idx = img_id;
        q_context.c2g.img_in = (uchar*)remote_dst;
        q_context.c2g.img_out = (uchar*)(connectionContext[0].output_addr + IMG_SZ * (img_id % OUTSTANDING_REQUESTS));

        rkey = connectionContext[1].input_rkey;
        post_rdma_write((uint64_t)&(queue_ptr->data[pi % NSLOTS]), sizeof(cpu_to_gpu_entry) , rkey, &q_context.c2g, mr_queue_context->lkey, 1, nullptr);
        
		//b. update pi
		q_context.pi = pi + 1;
        rkey = connectionContext[1].input_rkey;
		
        post_rdma_write((uint64_t)&(queue_ptr->pi), sizeof(int) , rkey, &q_context.pi, mr_queue_context->lkey, 2, nullptr);
        
		bool imgSent = false , entrySent = false , piSent = false;
        while (!imgSent || !entrySent || !piSent) {
            // Step 1: Poll for CQE
            struct ibv_wc wc;
            int ncqes = ibv_poll_cq(cq, 1, &wc);
            if (ncqes < 0) {
                perror("ibv_poll_cq() failed");
                exit(1);
            }
            if (ncqes > 0) {
		        VERBS_WC_CHECK(wc);
                if( wc.opcode == IBV_WC_RDMA_WRITE) {    
                    if(wc.wr_id == 0) imgSent = true;
                    if(wc.wr_id == 1) entrySent = true;
                    if(wc.wr_id == 2) piSent = true;
                }
                else{
                    printf("Unexpected completion\n");
                    assert(false);
                }
            }
        }
    }
	
	void read_image(queue<gpu_to_cpu_entry>* queue_ptr, int* img_id , int ci){

		//receive image id from queue
        uint32_t rkey = connectionContext[1].output_rkey;
		post_rdma_read(&(q_context.g2c), sizeof(gpu_to_cpu_entry) , mr_queue_context->lkey, (uint64_t)&(queue_ptr->data[ci]), rkey, 0);
		
		bool entryRead = false;
		while (!entryRead) {
            // Step 1: Poll for CQE
            struct ibv_wc wc;
            int ncqes = ibv_poll_cq(cq, 1, &wc);
            if (ncqes < 0) {
                perror("ibv_poll_cq() failed");
                exit(1);
            }
            if (ncqes > 0) {
		        VERBS_WC_CHECK(wc);
                if(wc.opcode == IBV_WC_RDMA_READ) {
                    if(wc.wr_id == 0) entryRead = true;
                } else {
                    printf("Unexpected completion\n");
                    assert(false);
                }
            }
        }
		
		*img_id = q_context.g2c.img_idx;

		//receive image from queue
        uint64_t remote_dst = connectionContext[0].output_addr + IMG_SZ * ((*img_id) % OUTSTANDING_REQUESTS);
        rkey = connectionContext[0].output_rkey;
        
        post_rdma_read(out_images + (*img_id) * IMG_SZ, IMG_SZ , mr_images_out->lkey, remote_dst, rkey, 1);
		
		//update ci
		q_context.ci = ci + 1;
        rkey = connectionContext[1].output_rkey;
        post_rdma_write((uint64_t)&(queue_ptr->ci), sizeof(int) , rkey, &(q_context.ci), mr_queue_context->lkey, 2);
		
        bool imgRead = false, ciSent = false;
        while (!imgRead || !ciSent) {
            // Step 1: Poll for CQE
            struct ibv_wc wc;
            int ncqes = ibv_poll_cq(cq, 1, &wc);
            if (ncqes < 0) {
                perror("ibv_poll_cq() failed");
                exit(1);
            }
            if (ncqes > 0) {
		        VERBS_WC_CHECK(wc);
                if(wc.opcode == IBV_WC_RDMA_READ) {
                    if(wc.wr_id == 1) imgRead = true;
                }
                else if(wc.opcode == IBV_WC_RDMA_WRITE) {
					if(wc.wr_id == 2) ciSent = true;
				} else {
                    printf("Unexpected completion\n");
                    assert(false);
                }
            }
        }
    }
};

std::unique_ptr<rdma_server_context> create_server(mode_enum mode, uint16_t tcp_port)
{
    switch (mode) {
    case MODE_RPC_SERVER:
        return std::make_unique<server_rpc_context>(tcp_port);
    case MODE_QUEUE:
        return std::make_unique<server_queues_context>(tcp_port);
    default:
        printf("Unknown mode.\n");
        exit(1);
    }
}

std::unique_ptr<rdma_client_context> create_client(mode_enum mode, uint16_t tcp_port)
{
    switch (mode) {
    case MODE_RPC_SERVER:
        return std::make_unique<client_rpc_context>(tcp_port);
    case MODE_QUEUE:
        return std::make_unique<client_queues_context>(tcp_port);
    default:
        printf("Unknown mode.\n");
        exit(1);
    }
}
